#include <hip/hip_runtime.h>

extern "C" {
  int cudaPyHostToDevice(void*, void*, size_t, size_t);
  int cudaPyDeviceToHost(void*, void*, size_t, size_t);
  void* cudaPyAllocArray(size_t, size_t);
  int cudaPyFree(void*);
}


int cudaPyHostToDevice(void* dst, void* src, size_t N, size_t tsize) {
  return hipMemcpy(dst, src, N * tsize, hipMemcpyHostToDevice);
}


int cudaPyDeviceToHost(void* dst, void* src, size_t N, size_t tsize) {
  return hipMemcpy(dst, src, N * tsize, hipMemcpyDeviceToHost);
}


void* cudaPyAllocArray(size_t N, size_t tsize) {
  void* arr;
  size_t arraySize = 2 * sizeof(size_t) + N * tsize;

  if (hipMalloc(&arr, arraySize))
    return NULL;
  hipMemset(&arr, 0, arraySize);

  size_t header[2] = {tsize, N};
  hipMemcpy(arr, &header, sizeof(size_t) * sizeof(header), hipMemcpyHostToDevice);

  return (void*)((size_t*)arr + 2);
}


int cudaPyFree(void* input) {
  return hipFree((void*)((size_t*)input - 2));
}
